#include "hip/hip_runtime.h"
#pragma once

#include <optix_world.h>

#include "random_lcg.cu"

#include "openmc/distribution.h"

using namespace openmc;

__device__ __forceinline__
float _sample_tabular(const Tabular_& t)
{
  // rtPrintf("Tabular_.x_ buffer id: %d\n", t.x_.getId());
  // rtPrintf("Tabular_.c_ buffer id: %d\n", t.c_.getId());
  // rtPrintf("Tabular_.p_ buffer id: %d\n", t.p_.getId());

  // Sample value of CDF
  float c = prn();

  // Find first CDF bin which is above the sampled value
  float c_i = t.c_[0];
  int i;
  size_t n = t.c_.size();
  for (i = 0; i < n - 1; ++i) {
    if (c <= t.c_[i+1]) break;
    c_i = t.c_[i+1];
  }

  // Determine bounding PDF values
  float x_i = t.x_[i];
  float p_i = t.p_[i];

  if (t.interp_ == Interpolation::histogram) {
    // Histogram interpolation
    if (p_i > 0.0f) {
      return x_i + (c - c_i)/p_i;
    } else {
      return x_i;
    }
  } else {
    // Linear-linear interpolation
    float x_i1 = t.x_[i + 1];
    float p_i1 = t.p_[i + 1];

    float m = (p_i1 - p_i)/(x_i1 - x_i);
    if (m == 0.0f) {
      return x_i + (c - c_i)/p_i;
    } else {
      return x_i + (sqrtf(fmaxf(0.0f, p_i*p_i + 2*m*(c - c_i))) - p_i)/m;
    }
  }
}